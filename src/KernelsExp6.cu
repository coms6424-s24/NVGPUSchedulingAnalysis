#include "hip/hip_runtime.h"
#include "../include/KernelsExp6.cuh"
#include <>
#include <hip/hip_math_constants.h>

const unsigned long long target_duration = 4000000000ULL; // 4e9 clock ticks ≈ 1 second

// Kernel 1: Performs trigonometric operations on a float value.
__global__ void kernel1()
{
    unsigned long long start_clock = clock64();
    unsigned long long end_clock = start_clock + target_duration;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float value = static_cast<float>(idx) * 1.5f;

    while (clock64() < end_clock) {
        value = value * value;
    }
}

__global__ void kernel2()
{
    const unsigned long long target_duration2 = 2000000000ULL; // 2e9 clock ticks ≈ 0.5 second

    unsigned long long start_clock = clock64();
    unsigned long long end_clock = start_clock + target_duration2;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float value = static_cast<float>(idx) * 1.5f;

    while (clock64() < end_clock) {
        value = value * value;
    }
}