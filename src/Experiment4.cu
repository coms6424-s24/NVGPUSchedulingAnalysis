// Shared Memory Testing

#include <Windows.h>
#include <chrono>
#include "../include/KernelsExp4.cuh"
#include "../include/StreamManager.hpp"
#include "../include/TimerSpin.cuh"
#include "../include/KernelConfig4.hpp"

KernelConfig g_kernel_config;

int main()
{
    int device;
    hipDeviceProp_t properties;
    hipGetDevice(&device);
    hipGetDeviceProperties(&properties, device);
    std::cout << "Device Number: " << device << std::endl
              << "  Device name: " << properties.name << std::endl
              << "  Number of multiprocessors: " << properties.multiProcessorCount << std::endl
              << "  Maximum number of threads per multiprocessor: " << properties.maxThreadsPerMultiProcessor << std::endl
              << "  Maximum number of warps per multiprocessor: " << properties.maxThreadsPerMultiProcessor / 32 << std::endl
              << "  Shared Memory per SM: " << properties.sharedMemPerMultiprocessor << " bytes" << std::endl
              << "  Shared Memory per Block: " << properties.sharedMemPerBlock << " bytes" << std::endl;

    g_kernel_config.SetParameter(properties);

    int data_size = 1024;
    size_t bytes = data_size * sizeof(float);

    float *input1, *output1;
    float *d_input1, *d_output1;

    float *input2, *output2;
    float *d_input2, *d_output2;

    input1 = new float[data_size];
    output1 = new float[data_size];

    input2 = new float[data_size];
    output2 = new float[data_size];

    for (int i = 0; i < data_size; ++i) {
        input1[i] = float(i);
        input2[i] = float(i);
    }

    hipMalloc(&d_input1, bytes);
    hipMalloc(&d_output1, bytes);

    hipMalloc(&d_input2, bytes);
    hipMalloc(&d_output2, bytes);

    StreamManager stream1;
    StreamManager stream2;

    std::cout << "Adding kernels to stream managers..." << std::endl;

    stream1.AddCopyOperation(d_input1, input1, bytes, hipMemcpyHostToDevice);
    stream2.AddCopyOperation(d_input2, input2, bytes, hipMemcpyHostToDevice);

    stream1.ExecuteScheduledOperations();
    stream2.ExecuteScheduledOperations();

    stream1.Synchronize();
    stream2.Synchronize();

    stream1.AddKernel("shared1", g_kernel_config.grid_size1, g_kernel_config.block_size1, g_kernel_config.data_size1, 
    MemoryIntensiveKernel, d_input1, g_kernel_config.block_size1.x * sizeof(float));

    stream2.AddKernel("shared2", g_kernel_config.grid_size2, g_kernel_config.block_size2, g_kernel_config.data_size2, 
    MemoryIntensiveKernel, d_input2, g_kernel_config.block_size2.x * sizeof(float));

    stream2.AddKernel("shared3", g_kernel_config.grid_size2, g_kernel_config.block_size2, 0, 
    MemoryIntensiveKernel, d_input2, 256);

    stream1.ScheduleKernelExecution("shared1");
    stream2.ScheduleKernelExecution("shared2");
    stream2.ScheduleKernelExecution("shared3");

    stream1.ExecuteScheduledOperations();
    stream2.ExecuteScheduledOperations();

    stream1.Synchronize();
    stream2.Synchronize();

    hipFree(d_input1);
    hipFree(d_output1);
    hipFree(d_input2);
    hipFree(d_output2);
    delete[] input1;
    delete[] output1;
    delete[] input2;
    delete[] output2;

    std::cout << "All operations completed." << std::endl;

    return 0;
}
