#include <Windows.h>
#include <chrono>
#include "../include/KernelsExp1.cuh"
#include "../include/StreamManager.hpp"
#include "../include/TimerSpin.cuh"
#include "../include/KernelConfig2.hpp"

KernelConfig g_kernel_config;

int main()
{
    int device;
    hipDeviceProp_t properties;
    hipGetDevice(&device);
    hipGetDeviceProperties(&properties, device);
    std::cout << "Device Number: " << device << std::endl
              << "  Device name: " << properties.name << std::endl
              << "  Number of multiprocessors: " << properties.multiProcessorCount << std::endl
              << "  Maximum number of threads per multiprocessor: " << properties.maxThreadsPerMultiProcessor << std::endl
              << "  Maximum number of warps per multiprocessor: " << properties.maxThreadsPerMultiProcessor / 32 << std::endl;

    g_kernel_config.SetParameter(properties);

    void* device_data = nullptr;
    void* host_data = nullptr;

    hipMalloc(&device_data, 256 * 1024 * 1024);
    host_data = malloc(256 * 1024 * 1024);

    StreamManager stream1;
    StreamManager stream2;
    StreamManager stream3;

    std::cout << "Adding kernels to stream managers..." << std::endl;

    // Add all kernels into the stream manager kernel map
    stream1.AddKernel("kernel1", g_kernel_config.grid_size1, g_kernel_config.block_size1, 0, kernel1);
    stream1.AddKernel("kernel2", g_kernel_config.grid_size2, g_kernel_config.block_size2, 0, kernel2);
    stream1.AddKernel("kernel3", g_kernel_config.grid_size3, g_kernel_config.block_size3, 0, kernel3);
    
    stream2.AddKernel("kernel4", g_kernel_config.grid_size4, g_kernel_config.block_size4, g_kernel_config.shared_mem_size4, kernel4);
    stream2.AddKernel("kernel6", g_kernel_config.grid_size6, g_kernel_config.block_size6, 0, kernel6);

    stream3.AddKernel("kernel5", g_kernel_config.grid_size5, g_kernel_config.block_size5, g_kernel_config.shared_mem_size5, kernel5);

    // Schedule and execute stored kernels
    // Stream1: kernel1, kernel2, kernel3
    stream1.ScheduleKernelExecution("kernel1");
    stream1.ScheduleKernelExecution("kernel2");
    stream1.AddCopyOperation(device_data, host_data, g_kernel_config.copy_size2, hipMemcpyDeviceToHost); // Copy after K2
    stream1.AddCopyOperation(host_data, device_data, g_kernel_config.copy_size3, hipMemcpyHostToDevice); // Copy before K3
    stream1.ScheduleKernelExecution("kernel3");
    stream1.AddCopyOperation(device_data, host_data, g_kernel_config.copy_size3, hipMemcpyDeviceToHost); // Copy after K3

    // Stream2: kernel4
    stream2.ScheduleKernelExecution("kernel4");

    std::cout << "Executing scheduled kernel1, 2, 3 on stream1, kernel4 one stream2..." << std::endl;
    stream1.ExecuteScheduledOperations();
    stream2.ExecuteScheduledOperations();

    stream1.Synchronize();
    stream2.Synchronize();
    std::cout << "kernel1, 2, 3 on stream1 and kernel4 one stream2 completed" << std::endl;

    int wait_time_ms2 = 400;
    std::cout << "Sleep waiting for "<< wait_time_ms2 << "ms" << std::endl;
    std::this_thread::sleep_for(std::chrono::milliseconds(wait_time_ms2));

    stream3.ScheduleKernelExecution("kernel5");
    stream3.AddCopyOperation(device_data, host_data, g_kernel_config.copy_size5, hipMemcpyDeviceToHost);

    std::cout << "Executing scheduled kernel5 on stream3..." << std::endl;
    
    stream3.ExecuteScheduledOperations();
    stream3.Synchronize();
    std::cout << "kernel5 on stream3 completed" << std::endl;

    int wait_time_ms3 = 1000;
    std::cout << "Sleep waiting for " << wait_time_ms3 <<"ms" << std::endl;
    std::this_thread::sleep_for(std::chrono::milliseconds(wait_time_ms3));

    stream2.ScheduleKernelExecution("kernel6");
    stream2.AddCopyOperation(device_data, host_data, g_kernel_config.copy_size6, hipMemcpyDeviceToHost);
    std::cout << "Executing kernel6 on stream2..." << std::endl;
    stream2.ExecuteScheduledOperations();
    stream2.Synchronize();
    std::cout << "Additional stream2 operations completed." << std::endl;

    std::cout << "All operations completed." << std::endl;

    return 0;
}
